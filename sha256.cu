
#include <hip/hip_runtime.h>
#include <cstring>
#include <fstream>
#include <iostream>
#ifndef SHA256_H
#define SHA256_H
#include <string>

class SHA256
{
protected:

    const static unsigned int sha256_k[];
    static const unsigned int SHA224_256_BLOCK_SIZE = (512/8);
public:
    void init();
    void update(const unsigned char *message, unsigned int len);
    void final(unsigned char *digest);
    static const unsigned int DIGEST_SIZE = ( 256 / 8);

protected:
    void transform(const unsigned char *message, unsigned int block_nb, unsigned int number_of_elements);
    unsigned int m_tot_len;
    unsigned int m_len;
    unsigned char m_block[2*SHA224_256_BLOCK_SIZE];
    unsigned int m_h[8];
};

std::string sha256(std::string input);

#define SHA2_SHFR(x, n)    (x >> n)
#define SHA2_ROTR(x, n)   ((x >> n) | (x << ((sizeof(x) << 3) - n)))
#define SHA2_ROTL(x, n)   ((x << n) | (x >> ((sizeof(x) << 3) - n)))
#define SHA2_CH(x, y, z)  ((x & y) ^ (~x & z))
#define SHA2_MAJ(x, y, z) ((x & y) ^ (x & z) ^ (y & z))
#define SHA256_F1(x) (SHA2_ROTR(x,  2) ^ SHA2_ROTR(x, 13) ^ SHA2_ROTR(x, 22))
#define SHA256_F2(x) (SHA2_ROTR(x,  6) ^ SHA2_ROTR(x, 11) ^ SHA2_ROTR(x, 25))
#define SHA256_F3(x) (SHA2_ROTR(x,  7) ^ SHA2_ROTR(x, 18) ^ SHA2_SHFR(x,  3))
#define SHA256_F4(x) (SHA2_ROTR(x, 17) ^ SHA2_ROTR(x, 19) ^ SHA2_SHFR(x, 10))
#define SHA2_UNPACK32(x, str)                         \
{                                                     \
    *((str) + 3) = (unsigned char) ((x)      );       \
    *((str) + 2) = (unsigned char) ((x) >>  8);       \
    *((str) + 1) = (unsigned char) ((x) >> 16);       \
    *((str) + 0) = (unsigned char) ((x) >> 24);       \
}
#define SHA2_PACK32(str, x)                           \
{                                                     \
    *(x) =   ((unsigned int) *((str) + 3)      )      \
           | ((unsigned int) *((str) + 2) <<  8)      \
           | ((unsigned int) *((str) + 1) << 16)      \
           | ((unsigned int) *((str) + 0) << 24);     \
}
#endif

using std::string;
using std::cout;
using std::endl;


const unsigned int SHA256::sha256_k[64] = //UL = uint32
            {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
             0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
             0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
             0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
             0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
             0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
             0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
             0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
             0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
             0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
             0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
             0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
             0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
             0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
             0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
             0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__device__ unsigned int t11;
__device__ unsigned int t12;

__global__ void kernel1(unsigned char* sub_block_d, unsigned int *w_d, unsigned int *wv, unsigned int *m_h) {

    int j = threadIdx.x;

    wv[j] = m_h[j];

    if ( j >= 8 ) {
        SHA2_PACK32(&sub_block_d[j << 2], &w_d[j]);
        // printf("%d %d\n", sub_block_d[j << 2], w_d[j]);
    }
}

__global__ void kernel2(unsigned int *w_d, unsigned int *wv, unsigned int *sha256_k) {

    int j = threadIdx.x;
    if (j >= 16){
        w_d[j] = SHA256_F4(w_d[j -  2]) + w_d[j -  7] + SHA256_F3(w_d[j - 15]) + w_d[j - 16];
    }

    t11 = wv[7] + SHA256_F2(wv[4]) + SHA2_CH(wv[4], wv[5], wv[6])
    + sha256_k[j] + w_d[j];
    t12 = SHA256_F1(wv[0]) + SHA2_MAJ(wv[0], wv[1], wv[2]);
    wv[7] = wv[6];
    wv[6] = wv[5];
    wv[5] = wv[4];
    wv[4] = wv[3] + t11;
    wv[3] = wv[2];
    wv[2] = wv[1];
    wv[1] = wv[0];
    wv[0] = t11 + t12;

}

__global__ void kernel3(unsigned int *wv, unsigned int *m_h) {

    int j = threadIdx.x;    
    m_h[j] += wv[j];

}

void SHA256::transform(const unsigned char *message, unsigned int block_nb, unsigned int number_of_elements)
{
    unsigned int w[64];
    unsigned int wv[8];
    unsigned int *wv1;
    unsigned int *w_d;
    unsigned int *sha256_k1;

    unsigned int *wv_d;
    unsigned int *m_h_d;

    const unsigned char *sub_block;
    unsigned char *sub_block_d;
    int i;
    for (i = 0; i < (int) block_nb; i++) {

        sub_block = message + (i << 6);

        ///////////////////////// K 1 ////////////////////////////////
        hipMalloc((void **) &sub_block_d, sizeof(unsigned char) * number_of_elements);
        hipMemcpy(sub_block_d, sub_block, sizeof(unsigned char) * number_of_elements, hipMemcpyHostToDevice);

        hipMalloc((void **) &w_d, sizeof(unsigned int) * 64);
        hipMemcpy(w_d, w, sizeof(unsigned int) * 64, hipMemcpyHostToDevice);

        hipMalloc((void **) &wv_d, sizeof(unsigned int) * 8);
        hipMemcpy(wv_d, wv, sizeof(unsigned int) * 8, hipMemcpyHostToDevice);

        hipMalloc((void **) &m_h_d, sizeof(unsigned int) * 8);
        hipMemcpy(m_h_d, m_h, sizeof(unsigned int) * 8, hipMemcpyHostToDevice);

        kernel1<<<1, 16>>>(sub_block_d, w_d, wv_d, m_h_d);

        hipDeviceSynchronize();
        hipMemcpy(w, w_d, sizeof(unsigned int) * 64, hipMemcpyDeviceToHost);
        hipMemcpy(wv, wv_d, sizeof(unsigned int) * 8, hipMemcpyDeviceToHost);
        hipMemcpy(m_h, m_h_d, sizeof(unsigned int) * 8, hipMemcpyDeviceToHost);
        // for(int k = 0; k < 64; k++){
        //     printf("%d ", w[k]);
        // }

        //////////////////////// K 2 ////////////////////////////////
        hipMemcpy(w_d, w, sizeof(unsigned int) * 64, hipMemcpyHostToDevice);

        hipMalloc((void **) &wv1, sizeof(unsigned int) * 8);
        hipMemcpy(wv1, wv, sizeof(unsigned int) * 8, hipMemcpyHostToDevice);

        hipMalloc((void **) &sha256_k1, sizeof(unsigned int) * 64);
        hipMemcpy(sha256_k1, sha256_k, sizeof(unsigned int) * 64, hipMemcpyHostToDevice);

        kernel2<<<1, 64>>>(w_d, wv1, sha256_k1);

        hipDeviceSynchronize();
        hipMemcpy(wv, wv1, sizeof(unsigned int) * 8, hipMemcpyDeviceToHost);
        hipMemcpy(w, w_d, sizeof(unsigned int) * 64, hipMemcpyDeviceToHost);
       
        //////////////////////////K 3///////////////////////////////
        hipMemcpy(wv1, wv, sizeof(unsigned int) * 8, hipMemcpyHostToDevice);
        hipMemcpy(m_h_d, m_h, sizeof(unsigned int) * 8, hipMemcpyHostToDevice);

        kernel3<<<1, 8>>>(wv1, m_h_d);

        hipDeviceSynchronize();
        hipMemcpy(wv, wv1, sizeof(unsigned int) * 8, hipMemcpyDeviceToHost);
        hipMemcpy(m_h, m_h_d, sizeof(unsigned int) * 8, hipMemcpyDeviceToHost);
    }
}

void SHA256::init()
{
    m_h[0] = 0x6a09e667;
    m_h[1] = 0xbb67ae85;
    m_h[2] = 0x3c6ef372;
    m_h[3] = 0xa54ff53a;
    m_h[4] = 0x510e527f;
    m_h[5] = 0x9b05688c;
    m_h[6] = 0x1f83d9ab;
    m_h[7] = 0x5be0cd19;
    m_len = 0;
    m_tot_len = 0;
}

void SHA256::update(const unsigned char *message, unsigned int len)
{
    unsigned int block_nb;
    unsigned int new_len, rem_len, tmp_len;
    const unsigned char *shifted_message;
    tmp_len = SHA224_256_BLOCK_SIZE - m_len;
    rem_len = len < tmp_len ? len : tmp_len;
    memcpy(&m_block[m_len], message, rem_len);
    if (m_len + len < SHA224_256_BLOCK_SIZE) {
        m_len += len;
        return;
    }
    new_len = len - rem_len;
    block_nb = new_len / SHA224_256_BLOCK_SIZE;
    shifted_message = message + rem_len;
    block_nb = new_len / SHA224_256_BLOCK_SIZE;
    transform(m_block, 1, SHA224_256_BLOCK_SIZE);
    transform(shifted_message, block_nb, SHA224_256_BLOCK_SIZE);
    rem_len = new_len % SHA224_256_BLOCK_SIZE;
    memcpy(m_block, &shifted_message[block_nb << 6], rem_len);
    m_len = rem_len;
    m_tot_len += (block_nb + 1) << 6;
}

void SHA256::final(unsigned char *digest)
{
    unsigned int block_nb;
    unsigned int pm_len;
    unsigned int len_b;
    int i;
    block_nb = (1 + ((SHA224_256_BLOCK_SIZE - 9)
                     < (m_len % SHA224_256_BLOCK_SIZE)));
    len_b = (m_tot_len + m_len) << 3;
    pm_len = block_nb << 6;
    memset(m_block + m_len, 0, pm_len - m_len);
    m_block[m_len] = 0x80;
    SHA2_UNPACK32(len_b, m_block + pm_len - 4);
    transform(m_block, block_nb, block_nb * len_b);
    for (i = 0 ; i < 8; i++) {
        SHA2_UNPACK32(m_h[i], &digest[i << 2]);
    }
}

std::string sha256(std::string input)
{
    unsigned char digest[SHA256::DIGEST_SIZE];
    memset(digest,0,SHA256::DIGEST_SIZE);

    SHA256 ctx = SHA256();
    ctx.init();
    ctx.update( (unsigned char*)input.c_str(), input.length());
    ctx.final(digest);

    char buf[2*SHA256::DIGEST_SIZE+1];
    buf[2*SHA256::DIGEST_SIZE] = 0;
    for (int i = 0; i < SHA256::DIGEST_SIZE; i++)
        sprintf(buf+i*2, "%02x", digest[i]);
    return std::string(buf);
}
int main(int argc, char *argv[])
{
    string input = "apple";
    string output1 = sha256(input);

    cout << "sha256('"<< input << "'):" << output1 << endl;
    return 0;
}